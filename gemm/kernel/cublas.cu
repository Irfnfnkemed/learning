#include "../include/hipblas.h"

__host__ hipblasStatus_t blas::gpuBlasSgemm(int m, int n, int k, const half *A,
                                           const half *B, half *O,
                                           const float alpha, const float beta,
                                           hipblasHandle_t handle) {
  // use clublas to compute
  return hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, A,
                      HIP_R_16F, k, B, HIP_R_16F, k, &beta, O, HIP_R_16F, n,
                      HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
